#include "hip/hip_runtime.h"
#include <vector_td_utilities.h>
#include "cuPartialDifferenceOperator.h"
#include "cudaDeviceManager.h"
using namespace Gadgetron;

static inline
void setup_grid( unsigned int number_of_elements, dim3 *blockDim, dim3* gridDim)
{
    int cur_device = cudaDeviceManager::Instance()->getCurrentDevice();
    //int maxGridDim = cudaDeviceManager::Instance()->max_griddim(cur_device);
    int maxBlockDim = cudaDeviceManager::Instance()->max_blockdim(cur_device);
    int maxGridDim = 65535;

    // The default one-dimensional block dimension is...
    *blockDim = dim3(256);
    *gridDim = dim3((number_of_elements+blockDim->x-1)/blockDim->x);

    // Extend block/grid dimensions if we exceeded the maximum grid dimension
    if( gridDim->x > maxGridDim){
        blockDim->x = maxBlockDim;
        gridDim->x = (number_of_elements+blockDim->x-1)/blockDim->x;
    }

    if( gridDim->x > maxGridDim ){
        gridDim->x = (unsigned int)std::floor(std::sqrt(float(number_of_elements)/float(blockDim->x)));
        unsigned int num_elements_1d = blockDim->x*gridDim->x;
        gridDim->y *= ((number_of_elements+num_elements_1d-1)/num_elements_1d);
    }

    if( gridDim->x > maxGridDim || gridDim->y > maxGridDim){
        // If this ever becomes an issue, there is an additional grid dimension to explore for compute models >= 2.0.
        throw cuda_error("setup_grid(): too many elements requested.");
    }
}

template<class T, unsigned int D, int SKIP, int DIM > __global__ static void partialDifferenceKernel(const T* __restrict__ in, T* __restrict__ out, vector_td<int,D> dims, bool accumulate){

    const int elements = prod(dims);
    const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < prod(dims) ){

        auto co = idx_to_co(idx,dims);
        //co[DIM] = (co[DIM]+SKIP+dims[DIM])%dims[DIM];
        auto val1 = in[co_to_idx(co,dims)];
        co[DIM] = (co[DIM]+SKIP+dims[DIM])%dims[DIM];
        auto val2 = in[co_to_idx(co,dims)];
        if (accumulate)
            out[idx] += (val2-val1);
        else
            out[idx] = (val2-val1);
    }


};


template<class T, unsigned int D, int SKIP, int DIM> static void partialDifference(cuNDArray<T>* in, cuNDArray<T>* out,
                                                                                   bool accumulate){

    auto dims = *in->get_dimensions();
    auto vdims = vector_td<int,D>(from_std_vector<size_t,D>(dims));

    const size_t elements_per_batch = prod(vdims);
    const size_t elements_total = in->get_number_of_elements();


    dim3 grid,block;

    setup_grid(elements_per_batch,&block,&grid);


    for (int i = 0; i < elements_total/elements_per_batch; i++){
        partialDifferenceKernel<T,D,SKIP,DIM><<<grid,block>>>(in->get_data_ptr()+i*elements_per_batch,
                out->get_data_ptr()+i*elements_per_batch, vdims,accumulate);
    }

}


template<class T, unsigned int D> void cuPartialDifferenceOperator<T,D>::mult_M(cuNDArray<T> *in, cuNDArray<T> *out,
                                                                                bool accumulate) {

    switch(dim) {
        case 0:
            partialDifference<T,D,1,0>(in,out,accumulate);
            break;
        case 1:
            partialDifference<T,D,1,1>(in,out,accumulate);
            break;
        case 2:
            partialDifference<T,D,1,2>(in,out,accumulate);
            break;
        case 3:
            partialDifference<T,D,1,3>(in,out,accumulate);
            break;
        default:
            throw std::runtime_error("Unsupported dimension");

    }

}

template<class T, unsigned int D> void cuPartialDifferenceOperator<T,D>::mult_MH(cuNDArray<T> *in, cuNDArray<T> *out,
                                                                                bool accumulate) {

    switch(dim) {
        case 0:
            partialDifference<T,D,-1,0>(in,out,accumulate);
            break;
        case 1:
            partialDifference<T,D,-1,1>(in,out,accumulate);
            break;
        case 2:
            partialDifference<T,D,-1,2>(in,out,accumulate);
            break;
        case 3:
            partialDifference<T,D,-1,3>(in,out,accumulate);
            break;
        default:
            throw std::runtime_error("Unsupported dimension");

    }

}

template class cuPartialDifferenceOperator<float,1>;
template class cuPartialDifferenceOperator<float,2>;
template class cuPartialDifferenceOperator<float,3>;
template class cuPartialDifferenceOperator<float,4>;