#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#define NLM_WINDOW_RADIUS   3
#define NLM_BLOCK_RADIUS    3

#define NLM_WINDOW_AREA     ( (2 * NLM_WINDOW_RADIUS + 1) * (2 * NLM_WINDOW_RADIUS + 1)*(2 * NLM_WINDOW_RADIUS + 1) )
#define INV_KNN_WINDOW_AREA ( 1.0f / (float)KNN_WINDOW_AREA )
#define INV_NLM_WINDOW_AREA ( 1.0f / (float)NLM_WINDOW_AREA )

#define NLM_WEIGHT_THRESHOLD    0.10f
#define NLM_LERP_THRESHOLD      0.10f

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 8

#include "nonlocalMeans.h"
using namespace Gadgetron;
texture<float, 3, hipReadModeElementType> nlmTex;

////////////////////////////////////////////////////////////////////////////////
// NLM kernel
////////////////////////////////////////////////////////////////////////////////
__global__ static void NLM(
    float *dst,
    int imageW,
    int imageH,
    int imageD,
    float Noise
)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    const int iz = blockDim.z * blockIdx.z + threadIdx.z;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;
    const float z = (float)iz + 0.5f;

    if (ix < imageW && iy < imageH && iz < imageD)
    {
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)
                for (float k = -NLM_WINDOW_RADIUS; k <= NLM_WINDOW_RADIUS; k++)
            {
                //Find color distance from (x, y) to (x + j, y + i)
                float weightIJK = 0;

                for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                    for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++)
                        for (float l = -NLM_BLOCK_RADIUS; l <= NLM_BLOCK_RADIUS; m++){
                            float diff =tex3D(nlmTex, x + j + m, y + i + n, z + l + k) -
                                        tex3D(nlmTex, x + m, y + n, z + k);
                            weightIJK += diff*diff;
                        }




                //Derive final weight from color and geometric distance
                weightIJK     = __expf(-(weightIJK * Noise + (i * i + j * j+k*k) * INV_NLM_WINDOW_AREA));

                //Accumulate (x + j, y + i) texel color with computed weight
                float IJK = tex3D(nlmTex, x + j, y + i,z+k);
                accum += IJK * weightIJK;


                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJK;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights
        sumWeights = 1.0f / sumWeights;
        accum *= sumWeights;


        dst[imageW * iy + ix+imageD*imageW*iz] = accum ;
    }
}




void nonlocal_means(
    cuNDArray<float> *input, cuNDArray<float> *output ,
    float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();
    hipExtent extent;
    extent.width = imageW;
    extent.height = imageH;
    extent.depth = imageD;

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.kind = hipMemcpyDeviceToDevice;
    cpy_params.extent = extent;

    hipArray *image_array;
    hipMalloc3DArray(&image_array, &channelDesc, extent);
    cpy_params.dstArray = image_array;
    cpy_params.srcPtr = make_hipPitchedPtr
            ((void *) input->get_data_ptr(), extent.width * sizeof(float), extent.width, extent.height);
    hipMemcpy3D(&cpy_params);

    hipBindTextureToArray(nlmTex, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y,BLOCKDIM_Z);

    dim3 grid((imageW+threads.x-1)/threads.x, (imageH+threads.y-1)/threads.y,(imageD+threads.z-1)/threads.z);

    NLM<<<grid, threads>>>(output->get_data_ptr(), imageW, imageH,imageD, Noise);
}

