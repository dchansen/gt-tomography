#include "hip/hip_runtime.h"
#include "proton_kernels.h"
#include "vector_td_utilities.h"

#include "cuNDArray.h"

#include <stdio.h>
#include "float3x3.h"

//TODO: Get rid of these defines.
#define INT_STEPS 2048
#define MAXSTEP 512
#define STEPS 3

using namespace Gadgetron;

/*template <typename T> __inline__ __host__ __device__ T sgn(T val)
{
    return copysign(T(1),val);
}


template< class T, unsigned int D > __inline__ __host__ __device__ vector_td<T,D> sgn ( const vector_td<T,D> &v1)
{
  vector_td<T,D> res;
  for(unsigned int i=0; i<D; i++ ) res.vec[i] =sgn(v1.vec[i]);
  return res;
}*/
/*
template< class T, class R, unsigned int D > __inline__ __host__ __device__ vector_td<typename vectorTDReturnType<T,R>::type,D> operator* ( const vector_td<T,D> &v1, const vector_td<R,D> &v2 )
{
  vector_td<typename vectorTDReturnType<T,R>::type,D> res;
  for(unsigned int i=0; i<D; i++ )  res.vec[i]=v1.vec[i]*v2.vec[i];
  return res;
}

template< class T, class R, unsigned int D > __inline__ __host__ __device__ vector_td<typename vectorTDReturnType<T,R>::type,D> operator/ ( const vector_td<T,D> &v1, const vector_td<R,D> &v2 )
{
  vector_td<typename vectorTDReturnType<T,R>::type,D> res;
  for(unsigned int i=0; i<D; i++ )  res.vec[i]=v1.vec[i]/v2.vec[i];
  return res;
}
 */
template< class T, unsigned int D > __inline__ __host__ __device__ vector_td<T,D> remove_neg ( const vector_td<T,D> &v1)
																		{
	vector_td<T,D> res;
	for(unsigned int i=0; i<D; i++ ){
		if (v1.vec[i]<0){ res.vec[i] =0;}
		else {res.vec[i]=v1.vec[i];}
	}

	return res;
																		}



template <class REAL, class OP> __global__ void Gadgetron::path_kernel2(OP op, const vector_td<REAL,3> * __restrict__ splines,  const REAL* __restrict__ space_lengths, const vector_td<REAL,3> dims,
		const intd3 ndims, const int proj_dim, const int offset){

	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	if (idx < proj_dim){
		const int sid = idx*4;
		vector_td<int,3> co;
		op.begin(idx);

		int id,id_old;
		REAL t;
		//REAL length = lengths[idx];
		//REAL length = lengths[idx];
		REAL length=0;

		//Load in points to registers
		vector_td<REAL,3> p0 = splines[sid]; //Position at entrance
		vector_td<REAL,3> p1 = splines[sid+1]; // Position at exit
		vector_td<REAL,3> m0 = splines[sid+2]; // Direction at entrance
		vector_td<REAL,3> m1 = splines[sid+3]; // Direction at exit
		const REAL space_length0 = space_lengths[idx*2];
		const REAL space_length1 = space_lengths[idx*2+1];
		//Normalize
		m0 /= norm(m0);
		m1 /= norm(m1);

		vector_td<REAL,3> p;
		vector_td<REAL,3> p_old=p0;
		co = vector_td<int,3>((p_old+dims/2)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		const int lin_steps =max(ndims);
		for  (int i = 1; i < lin_steps; i++){
			t = REAL(i)/lin_steps;

			p = p0+m0*t*space_length0;
			co = vector_td<int,3>((p+dims/2)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(vector_td<int,3>(co),ndims);
			length += norm(p-p_old)/2;
			if(id_old != id){
				op(id_old,length);
				length=0;
			}
			length+= norm(p-p_old)/2;
			id_old=id;
			p_old=p;
		}
		p0 = p;


		op(id_old,length);

		length = 0;
		p_old = p1;
		co = vector_td<int,3>((p_old+dims/2)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		for  (int i = 1; i < lin_steps; i++){
			t = REAL(i)/(lin_steps);
			p = p1-m1*t*space_length1;
			co = vector_td<int,3>((p+dims/2)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(vector_td<int,3>(co),ndims);
			length += norm(p-p_old)/2;
			if(id_old != id){
				op(id_old,length);
				length=0;
			}
			length+= norm(p-p_old)/2;
			id_old=id;
			p_old=p;
		}

		p1 = p;

		op(id_old,length);
		length = 0;
		m0 *= norm(p0-p1);
		m1 *= norm(p0-p1);



		const vector_td<REAL,3> a = 2*p0+m0+m1-2*p1;
		const vector_td<REAL,3> b = -3*p0-2*m0+3*p1-m1;
		const vector_td<REAL,3> c = m0;
		const vector_td<REAL,3> d = p0;

		p_old = p0;
		co = vector_td<int,3>((p_old+dims/2)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		const int steps =max(ndims)*STEPS;
		for (int i = 1; i <= steps; i++){
			t = REAL(i)/(steps);
			p = d+t*(c+t*(b+t*a));

			//co = to_intd((p+dims/2)*ndims/dims);

			co = vector_td<int,3>((p+dims/2)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(vector_td<int,3>(co),ndims);
			//id=co_to_idx(co,ndims);
			//REAL step_length = norm((-1.0/(steps*steps*steps)-3*t*t/steps+3*t/(steps*steps))*a+(1.0/(steps*steps)-2*t/steps)*b-c/steps);
			length += norm(p-p_old)/2;

			if(id_old != id){
				//if (min(co) >= 0 && co < ndims ) res+=image[id_old]*length;
				op(id_old,length);
				length=0;
			}

			length+= norm(p-p_old)/2;
			id_old=id;
			p_old=p;
			//co = to_intd((p+dims/2)*ndims/dims);
			//co = amax(amin(co,ndims-1),0);
		}

		op.final(idx);
	}

}



template <class REAL, class OP> __global__ void Gadgetron::path_kernel(OP op,
		const vector_td<REAL,3> * __restrict__ splines,  const vector_td<REAL,3> dims,
		const typename intd<3>::Type ndims, const int proj_dim, const int offset){

	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	if (idx < proj_dim){
		const int sid = idx*4;
		vector_td<int,3> co;


		int id,id_old;
		REAL t;
		op.begin(idx);
		//REAL length = lengths[idx];
		//REAL length = lengths[idx];
		REAL length=0;

		//Load in points to registers
		const vector_td<REAL,3> p0 = splines[sid]; //Position at entrance
		const vector_td<REAL,3> p1 = splines[sid+1]; // Position at exit
		const vector_td<REAL,3> m0 = splines[sid+2]; // Direction at entrance
		const vector_td<REAL,3> m1 = splines[sid+3]; // Direction at exit


		const vector_td<REAL,3> a = 2*p0+m0+m1-2*p1;
		const vector_td<REAL,3> b = -3*p0-2*m0+3*p1-m1;
		const vector_td<REAL,3> c = m0;
		const vector_td<REAL,3> d = p0;


		vector_td<REAL,3> p;
		vector_td<REAL,3> p_old=d;
		co = vector_td<int,3>((p_old+dims/2)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		int steps =max(ndims)*STEPS;
		for (int i = 1; i < steps+1; i++){
			t = REAL(i)/(steps);
			p = d+t*(c+t*(b+t*a));

			//co = to_intd((p+dims/2)*ndims/dims);

			co = vector_td<int,3>((p+dims/2)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(vector_td<int,3>(co),ndims);
			//id=co_to_idx(co,ndims);
			//REAL step_length = norm((-1.0/(steps*steps*steps)-3*t*t/steps+3*t/(steps*steps))*a+(1.0/(steps*steps)-2*t/steps)*b-c/steps);
			length += norm(p-p_old)/2;

			if(id_old != id){
				//if (min(co) >= 0 && co < ndims ) res+=image[id_old]*length;
				op(id_old,length);
				length=0;
			}

			length+= norm(p-p_old)/2;
			id_old=id;
			p_old=p;
			//co = to_intd((p+dims/2)*ndims/dims);
			//co = amax(amin(co,ndims-1),0);
		}
		op.final(idx);
	}

}



template <class REAL> __global__ void Gadgetron::space_carver_kernel(const REAL* __restrict__ projections, REAL* __restrict__ image,
		const vector_td<REAL,3> * __restrict__ splines,   const vector_td<REAL,3> dims, REAL cutoff,
		const typename intd<3>::Type ndims, const int proj_dim, const int offset){

	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	if (idx < proj_dim){
		REAL proj = projections[idx];

		//Only use projections below the cutoff
		if (proj <= cutoff){
			const int sid = idx*4;
			vector_td<int,3> co;
			const vector_td<REAL,3> half_dims = dims/((REAL)2.0);
			int id,id_old;
			REAL t;

			//Load in points to registers
			vector_td<REAL,3> p0 = splines[sid];
			vector_td<REAL,3> p1 = splines[sid+1];
			vector_td<REAL,3> m0 = splines[sid+2];
			vector_td<REAL,3> m1 = splines[sid+3];
			REAL length = norm(p1-p0);
			m0 *= length/norm(m0);
			m1 *= length/norm(m1);

			vector_td<REAL,3> p;
			vector_td<REAL,3> p_old=p0+half_dims;
			co = vector_td<int,3>((p_old)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id_old=co_to_idx(co,ndims);

			int steps =max(ndims)*STEPS;
			for (int i = 1; i < steps; i++){

				t = REAL(i)/(steps);
				p=t*m0+p0+half_dims;
				co = vector_td<int,3>(p*ndims/dims);
				co = amax(amin(co,ndims-1),0);
				id=co_to_idx(co,ndims);
				//REAL step_length = norm(((dt*dt*dt)+3*t*t*dt+3*t*dt*dt)*a+(dt*dt+2*t*dt)*b+c*dt);


				if(id_old != id){
					//Set to 0.Ignore collisions.
					image[id_old]=0;
					//atomicExch(&(image[id_old]),REAL());
					//atomicCAS(&(image[id_old]),1,0);
				}
				id_old=id;
				p_old=p;
				//co = to_intd((p+dims/2)*ndims/dims);
				//co = amax(amin(co,ndims-1),0);
			}
		}

	}

}


template<class REAL> __global__ void Gadgetron::rotate_splines_kernel(vector_td<REAL,3> * splines,REAL angle, unsigned int total, unsigned int offset){
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;

	if (idx < total){
		const unsigned int sid = idx*4;
		const float3x3 inverseRotation = calcRotationMatrixAroundZ(-angle);
		vector_td<REAL,3> p0 = mul(inverseRotation,splines[sid]);
		vector_td<REAL,3> p1 = mul(inverseRotation,splines[sid+1]);
		vector_td<REAL,3> m0 = mul(inverseRotation,splines[sid+2]);
		vector_td<REAL,3> m1 = mul(inverseRotation,splines[sid+3]);

		splines[sid] = p0;
		splines[sid+1] = p1;
		splines[sid+2] = m0;
		splines[sid+3] = m1;
	}
}

template <class REAL> __global__ void Gadgetron::crop_splines_kernel(vector_td<REAL,3> * splines, REAL* projections, const  vector_td<REAL,3>  dims, const int proj_dim,const REAL background,int offset)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;

	if (idx < proj_dim){
		const int sid = idx*4;

		const vector_td<REAL,3> half_dims = dims/((REAL)2.0);



		REAL t,told;

		//Load in points to registers
		vector_td<REAL,3> p0 = splines[sid];
		vector_td<REAL,3> p1 = splines[sid+1];
		vector_td<REAL,3> m0 = splines[sid+2];
		vector_td<REAL,3> m1 = splines[sid+3];

		REAL length = norm(p1-p0);
		m0 *= length/norm(m0);
		m1 *= length/norm(m1);
		vector_td<REAL,3> p,pt0,pt1;

		t=0;
		for (int i = 0; i < MAXSTEP; i++){
			told = t;
			t = ((REAL) i)/MAXSTEP;
			//t2 = t*t;

			//p = (2*t3-3*t2+1)*p0+(t3-2*t2+t)*m0+(3*t2-2*t3)*p1+(t3-t2)*m1+half_dims;
			p=t*m0+p0+half_dims;

			if ( min(p) >= 0 && p < dims) break;

		}
		t = told;
		//t2 = t*t;

		//pt0 =  (2*t3-3*t2+1)*p0+(t3-2*t2+t)*m0+(3*t2-2*t3)*p1+(t3-t2)*m1; //Calculate new starting point
		pt0=t*m0+p0;

		t = 0;
		for (int i = 0; i < MAXSTEP; i++){
			told = t;
			t = ((REAL) i)/MAXSTEP;
			//t2 = t*t;

			//p = (2*t3-3*t2+1)*p0+(t3-2*t2+t)*m0+(3*t2-2*t3)*p1+(t3-t2)*m1+half_dims;
			p=p1-t*m1+half_dims;
			if ( min(p) >= 0 && p < dims) break;

		}
		t = told;

		pt1=p1-t*m1;
		REAL deltaLength = norm(p1-pt1)+norm(p0-pt0);
		projections[idx] -= deltaLength*background;
		splines[sid]=pt0;
		splines[sid+1]=pt1;


	}
}

template <class REAL> __global__ void Gadgetron::crop_splines_hull_kernel(vector_td<REAL,3> * splines, REAL* projections,REAL* hull_mask,const vector_td<int,3> ndims, const  vector_td<REAL,3>  dims, const int proj_dim,const REAL background,int offset)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;

	if (idx < proj_dim){
		const int sid = idx*4;

		const vector_td<REAL,3> half_dims = dims/((REAL)2.0);


		vector_td<int,3> co;
		vector_td<REAL,3> pt0, pt1;
		int id, id_old;
		REAL t,told;
		//Load in points to registers
		vector_td<REAL,3> p0 = splines[sid];
		vector_td<REAL,3> p1 = splines[sid+1];
		vector_td<REAL,3> m0 = splines[sid+2];
		vector_td<REAL,3> m1 = splines[sid+3];
		REAL length = norm(p1-p0);
		m0 *= length/norm(m0);
		m1 *= length/norm(m1);

		vector_td<REAL,3> p;
		vector_td<REAL,3> p_old=p0+half_dims;
		co = vector_td<int,3>((p_old)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		int steps =max(ndims)*STEPS;
		t = 0;
		for (int i = 1; i <= steps; i++){
			told = t;
			t = REAL(i)/(steps);
			p=t*m0+p0+half_dims;
			co = vector_td<int,3>(p*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(co,ndims);
			//REAL step_length = norm(((dt*dt*dt)+3*t*t*dt+3*t*dt*dt)*a+(dt*dt+2*t*dt)*b+c*dt);
			if(id_old != id){

				if (hull_mask[id] > 0) break;
				//atomicCAS(&(image[id_old]),1,0);
			}
			id_old=id;
			p_old=p;
			//co = to_intd((p+dims/2)*ndims/dims);
			//co = amax(amin(co,ndims-1),0);
		}
		const REAL t1 = t;
		//t2 = t*t;

		//pt0 =  (2*t3-3*t2+1)*p0+(t3-2*t2+t)*m0+(3*t2-2*t3)*p1+(t3-t2)*m1; //Calculate new starting point
		pt0=told*m0+p0;

		t = 0;
		for (int i = 0; i <= steps; i++){
			told = t;
			t = ((REAL) i)/MAXSTEP;
			//t2 = t*t;

			//p = (2*t3-3*t2+1)*p0+(t3-2*t2+t)*m0+(3*t2-2*t3)*p1+(t3-t2)*m1+half_dims;
			p=p1-t*m1+half_dims;
			co = vector_td<int,3>(p*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(co,ndims);
			//REAL step_length = norm(((dt*dt*dt)+3*t*t*dt+3*t*dt*dt)*a+(dt*dt+2*t*dt)*b+c*dt);
			if(id_old != id){
				if (hull_mask[id] > 0) break;
				//atomicCAS(&(image[id_old]),1,0);
			}
			id_old=id;
			p_old=p;
		}


		pt1=p1-told*m1;
		if (t >= 1 || t1 >= 1){
			projections[idx]=0;
		} else {
			REAL deltaLength = norm(p1-pt1)+norm(p0-pt0);
			projections[idx] -= deltaLength*background;
		}
		splines[sid]=pt0;
		splines[sid+1]=pt1;


	}
}



template <class REAL> __global__ void Gadgetron::calc_spaceLengths_kernel(const vector_td<REAL,3> * __restrict__ splines, REAL* __restrict__ space_lengths,const REAL*  __restrict__ hull_mask,const vector_td<int,3> ndims, const  vector_td<REAL,3>  dims, const int proj_dim,int offset)
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;

	if (idx < proj_dim){
		const int sid = idx*4;

		const vector_td<REAL,3> half_dims = dims/((REAL)2.0);


		vector_td<int,3> co;
		vector_td<REAL,3> pt0, pt1;
		int id, id_old;
		REAL t;
		//Load in points to registers
		const vector_td<REAL,3> p0 = splines[sid];
		const vector_td<REAL,3> p1 = splines[sid+1];
		vector_td<REAL,3> m0 = splines[sid+2];
		vector_td<REAL,3> m1 = splines[sid+3];
		REAL length = norm(p1-p0);
		m0 *= length/norm(m0);
		m1 *= length/norm(m1);

		vector_td<REAL,3> p;

		co = vector_td<int,3>((p0+half_dims)*ndims/dims);
		co = amax(amin(co,ndims-1),0);
		id_old=co_to_idx(co,ndims);

		//Find first straight-line step
		int steps =max(ndims)*STEPS;
		t = 0;
		int i;
		for (i = 1; i <= steps; i++){
			t = REAL(i)/(steps);
			p=t*m0+p0;
			co = vector_td<int,3>((p+half_dims)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(co,ndims);
			if(id_old != id){
				if (hull_mask[id] > 0) break;
			}
			id_old=id;
		}
		if  (i == steps)space_lengths[idx*2] = 0; //hull is never hit
		else	space_lengths[idx*2] = norm(p-p0);


		//Find second straight-line step
		t = 0;
		for (i = 0; i <= steps; i++){
			t = ((REAL) i)/steps;
			p=p1-t*m1;
			co = vector_td<int,3>((p+half_dims)*ndims/dims);
			co = amax(amin(co,ndims-1),0);
			id=co_to_idx(co,ndims);
			if(id_old != id){
				if (hull_mask[id] > 0) break;
			}
			id_old=id;
		}
		if  (i == steps)space_lengths[idx*2+1] = 0; //hull is never hit
		else	space_lengths[idx*2+1] = norm(p-p1);
	}
}


template <class REAL> __global__ void Gadgetron::rescale_directions_kernel(vector_td<REAL,3> * splines, REAL* projections, const  vector_td<REAL,3>  dims,  const int proj_dim, const int offset )
{
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;

	if (idx < proj_dim){
		const int sid = idx*4;

		//Load in points to registers
		vector_td<REAL,3> p0 = splines[sid];
		vector_td<REAL,3> p1 = splines[sid+1];
		vector_td<REAL,3> m0 = splines[sid+2];
		vector_td<REAL,3> m1 = splines[sid+3];

		m0 /= norm(m0);
		m1 /= norm(m1);
		REAL length = norm(p1-p0);


		splines[sid+2]=m0*length;
		splines[sid+3]=m1*length;

	}
}

template <class REAL> __global__ void Gadgetron::points_to_coefficients(vector_td<REAL,3> * splines, int dim,int offset)
{


	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	const int sid = 4*idx;
	if (idx < dim){

		//Load in points to registers
		vector_td<REAL,3> p0 = splines[sid]; //Position at entrance
		vector_td<REAL,3> p1 = splines[sid+1]; // Position at exit
		vector_td<REAL,3> m0 = splines[sid+2]; // Direction at entrance
		vector_td<REAL,3> m1 = splines[sid+3]; // Direction at exit


		vector_td<REAL,3> a = 2*p0+m0+m1-2*p1;
		vector_td<REAL,3> b = -3*p0-2*m0+3*p1-m1;
		vector_td<REAL,3> c = m0;
		vector_td<REAL,3> d = p0;
		splines[sid]=a;
		splines[sid+1]=b;
		splines[sid+2]=c;
		splines[sid+3]=d;


	}

}

template <class REAL> __global__ void Gadgetron::spline_trapz_kernel(vector_td<REAL,3> * splines, REAL* lengths, int dim, int offset)
{
	//Admiral Ackbarz
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	const int sid = 4*idx;
	if (idx < dim){
		REAL res = 0;
		REAL s1;
		//Load in points to registers
		vector_td<REAL,3> a = splines[sid];
		vector_td<REAL,3> b = splines[sid+1];
		vector_td<REAL,3> c = splines[sid+2];
		vector_td<REAL,3> d = splines[sid+3];

		REAL t = 0;
		REAL s0 = norm(d);

		for (int i = 1; i < INT_STEPS; i++){
			t = ((REAL) i)/INT_STEPS;
			s1 = norm(c+t*(2*b+t*3*a));
			res += (s0+s1)/(2*INT_STEPS);
			s0 = s1;
		}
		lengths[idx]=res;
	}

}

/***
 * The Hansen correctional facility for young cubic splines.
 * Corrects the path length with the ratio between the length of the straight line approximation and the length of the spline
 */
template <class REAL> __global__ void Gadgetron::length_correction_kernel(vector_td<REAL,3> * splines, REAL* projections, int dim, int offset)
{
	//Admiral Ackbarz
	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	const int sid = 4*idx;
	if (idx < dim){
		REAL res = 0;
		REAL s1;
		//Load in points to registers
		vector_td<REAL,3> a = splines[sid];
		vector_td<REAL,3> b = splines[sid+1];
		vector_td<REAL,3> c = splines[sid+2];
		vector_td<REAL,3> d = splines[sid+3];

		REAL t = 0;

		REAL s0 = norm(d);

		for (int i = 1; i < INT_STEPS; i++){
			t = ((REAL) i)/INT_STEPS;
			s1 = norm(c+t*(2*b+t*3*a));
			res += (s0+s1)/(2*INT_STEPS);
			s0 = s1;
		}
		REAL tmp = norm(a+b+c)/res;
		projections[idx] *= tmp;
		if (idx-offset == 0)	printf("Correction: %f\n",tmp);
	}

}


template <class REAL> __global__ void Gadgetron::move_origin_kernel(vector_td<REAL,3> * splines,  const vector_td<REAL,3> origin,const unsigned int proj_dim, const unsigned int offset){

	const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x+offset;
	if (idx < proj_dim){
		const int sid = idx*4;

		splines[sid] -= origin;
		splines[sid+1] -= origin;
	}

}




template __global__ void Gadgetron::path_kernel<float, forward_functor<float> >(forward_functor<float>, const vector_td<float,3>  * __restrict__ , const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );
template __global__ void Gadgetron::path_kernel<float, backward_functor<float> >(backward_functor<float>, const vector_td<float,3>  * __restrict__ , const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );



template __global__ void Gadgetron::path_kernel<float, forward_norm_functor<float> >(forward_norm_functor<float>, const vector_td<float,3>  * __restrict__ , const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );

template __global__ void Gadgetron::path_kernel<float, backward_counting_functor<float> >(backward_counting_functor<float>, const vector_td<float,3>  * __restrict__ , const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );
template __global__ void Gadgetron::path_kernel2<float, forward_functor<float> >(forward_functor<float>, const vector_td<float,3>  * __restrict__ , const float * __restrict__ ,  const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );

template __global__ void Gadgetron::path_kernel2<float, forward_norm_functor<float> >(forward_norm_functor<float>, const vector_td<float,3>  * __restrict__ , const float * __restrict__ ,  const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );

template __global__ void Gadgetron::path_kernel2<float, backward_counting_functor<float> >(backward_counting_functor<float>, const vector_td<float,3>  * __restrict__ , const float * __restrict__ ,  const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );



template __global__ void Gadgetron::path_kernel2<float, backward_functor<float> >(backward_functor<float>, const vector_td<float,3>  * __restrict__ , const float * __restrict__ ,  const vector_td<float,3> ,
		const typename intd<3>::Type, const int , const int );

template __global__ void Gadgetron::space_carver_kernel<float>(const float* __restrict__ projections, float* __restrict__ image,
		const vector_td<float,3> * __restrict__ splines, const vector_td<float,3> dims, float cutoff,
		const typename intd<3>::Type ndims, const int proj_dim, const int offset);

template __global__ void Gadgetron::crop_splines_kernel<float>(vector_td<float,3> * splines, float* projections, const  vector_td<float,3>  dims, const int proj_dim,float background,int offset);
template __global__ void Gadgetron::rescale_directions_kernel<float>(vector_td<float,3> * splines, float* projections, const  vector_td<float,3>  dims,  const int proj_dim, const int offset);

template __global__ void Gadgetron::crop_splines_hull_kernel<float>(vector_td<float,3> * splines, float* projections,float* hull_mask,const vector_td<int,3,> ndims, const  vector_td<float,3>  dims, const int proj_dim,const float background,int offset);


template __global__ void Gadgetron::calc_spaceLengths_kernel(const vector_td<float,3> * __restrict__ splines, float* __restrict__ space_lengths,const float*  __restrict__ hull_mask,const vector_td<int,3> ndims, const  vector_td<float,3>  dims, const int proj_dim,int offset);
template __global__ void Gadgetron::points_to_coefficients<float>(vector_td<float,3> * splines, int dim,int offset);

template __global__  void Gadgetron::length_correction_kernel<float>(vector_td<float,3> * splines, float* projections, int dim, int offset);

template __global__ void Gadgetron::rotate_splines_kernel(vector_td<float,3> * splines,float angle, unsigned int total, unsigned int offset);
template __global__ void Gadgetron::move_origin_kernel(vector_td<float,3> * splines,const vector_td<float,3> origin, unsigned int total, unsigned int offset);
/*
template<> __global__ void forward_kernel<float>(float* image, float* projections,
		vector_td<float,3> * splines,  const vector_td<float,3> dims,
		const typename uintd<3>::Type ndims, const int proj_dim, const int offset);
 */
/*


template<> __global__ void backwards_kernel<float>;
template<> __global__ void rescale_splines_kernel<float>;

template<> __global__ void points_to_coefficients<float>;

template<> __global__ void spline_trapz_kernel<float>;
 */
