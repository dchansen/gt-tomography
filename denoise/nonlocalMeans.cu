#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#define NLM_WINDOW_RADIUS   5
#define NLM_BLOCK_RADIUS    3

#define NLM_WINDOW_AREA     ( (2 * NLM_WINDOW_RADIUS + 1) * (2 * NLM_WINDOW_RADIUS + 1)*(2 * NLM_WINDOW_RADIUS + 1) )
#define NLM_WINDOW_AREA2D     ( (2 * NLM_WINDOW_RADIUS + 1) * (2 * NLM_WINDOW_RADIUS + 1) )

#define INV_NLM_WINDOW_AREA ( 1.0f / (float)NLM_WINDOW_AREA )
#define INV_NLM_WINDOW_AREA2D ( 1.0f / (float)NLM_WINDOW_AREA2D )

#define NLM_WEIGHT_THRESHOLD    0.10f
#define NLM_LERP_THRESHOLD      0.10f

#define BLOCKDIM_X 8
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 8
#define BLOCKSTEP 4

#include "nonlocalMeans.h"
#include "cuNDArray_math.h"
#include <hipcub/hipcub.hpp>

using namespace Gadgetron;
texture<float, 3, hipReadModeElementType> nlmTex;

texture<float, 2, hipReadModeElementType> nlmTex2D;


__inline__ __device__
float warpReduceSum(float val) {
    for (int offset = warpSize/2; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}

__inline__ __device__
float blockReduceSum(float val) {

    typedef hipcub::BlockReduce<float,BLOCKDIM_X,hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,BLOCKDIM_Y,BLOCKDIM_Z> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    __shared__ float shares;
    float output = BlockReduce(temp_storage).Sum(val);

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) shares = output;
    __syncthreads();
    return shares;


/*
    int idx = threadIdx.x+threadIdx.y*BLOCKDIM_X+threadIdx.z*BLOCKDIM_X*BLOCKDIM_Y;
    int nthreads = BLOCKDIM_X*BLOCKDIM_Y*BLOCKDIM_Z;
    static __shared__ float shared[32]; // Shared mem for 32 partial sums


    int lane = idx % warpSize;
    int wid = idx / warpSize;

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (idx < nthreads / warpSize) ? shared[lane] : 0;

    if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

    if (idx == 0 ) shared[0] = val;
    __syncthreads();




    return shared[0];
    */

}

////////////////////////////////////////////////////////////////////////////////
// NLM kernel
////////////////////////////////////////////////////////////////////////////////
__global__ static void NLM3DBLOCK(
        float *dst,
        int imageW,
        int imageH,
        int imageD,
        int offset,
        float Noise
)
{





    const int ixo = blockDim.x * blockIdx.x + threadIdx.x;
    const int iyo = blockDim.y * blockIdx.y + threadIdx.y;
    const int izo = blockDim.z * blockIdx.z + threadIdx.z;
    //Add half of a texel to always address exact texel centers




    if (ixo < imageW && iyo < imageH && izo < imageD)
    {
        const int ix = (ixo+offset)%imageW;
        const int iy = (iyo+offset)%imageH;
        const int iz = (izo+offset)%imageD;
        const float x = (float)ix + 0.5f;
        const float y = (float)iy + 0.5f;
        const float z = (float)iz + 0.5f;
        float pixel =tex3D(nlmTex, x, y, z );
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)
                for (float k = -NLM_WINDOW_RADIUS; k <= NLM_WINDOW_RADIUS; k++)
                {
                    float  IJK = tex3D(nlmTex, x + j, y + i, z + k);
                    float diff = IJK-pixel;

                    float weightIJK = blockReduceSum(diff*diff);
                    //Derive final weight from color and geometric distance
                    weightIJK     = expf(-(weightIJK * Noise + (i * i + j * j+k*k) * INV_NLM_WINDOW_AREA));
                    //weightIJK     = expf(-(weightIJK * Noise ));

                    accum += IJK * weightIJK;


                    //Sum of weights for color normalization to [0..1] range
                    sumWeights  += weightIJK;

                    //Update weight counter, if NLM weight for current window texel
                    //exceeds the weight threshold
                    //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
                }

        //Normalize result color by sum of weights

        accum /= sumWeights;

        //if (fCount > NLM_LERP_THRESHOLD)
        dst[imageW * iy + ix+imageH*imageW*iz] = accum ;
    }
}

__global__ static void NLM3D(
    float *dst,
    int imageW,
    int imageH,
    int imageD,
    float Noise
)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    const int iz = blockDim.z * blockIdx.z + threadIdx.z;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;
    const float z = (float)iz + 0.5f;

    if (ix < imageW && iy < imageH && iz < imageD)
    {
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)
                for (float k = -NLM_WINDOW_RADIUS; k <= NLM_WINDOW_RADIUS; k++)
            {
                //Find color distance from (x, y) to (x + j, y + i)
                float weightIJK = 0;

                for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                    for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++)
                        for (float l = -NLM_BLOCK_RADIUS; l <= NLM_BLOCK_RADIUS; l++){
                            float diff =tex3D(nlmTex, x + j + m, y + i + n, z + l + k) -
                                        tex3D(nlmTex, x + m, y + n, z + l);
                            weightIJK += diff*diff;
                        }




                //Derive final weight from color and geometric distance
                weightIJK     = expf(-(weightIJK * Noise + (i * i + j * j+k*k) * INV_NLM_WINDOW_AREA));
                //weightIJK     = expf(-(weightIJK * Noise ));




                //Accumulate (x + j, y + i) texel color with computed weight#
                float IJK = tex3D(nlmTex, x + j, y + i,z+k);
                accum += IJK * weightIJK;


                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJK;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights

        accum /= sumWeights;

        //if (fCount > NLM_LERP_THRESHOLD)
        dst[imageW * iy + ix+imageH*imageW*iz] = accum ;
    }
}


__global__ static void NLM3DPoisson(
        float *dst,
        int imageW,
        int imageH,
        int imageD,
        float Noise
)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;
    const int iz = blockDim.z * blockIdx.z + threadIdx.z;
    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;
    const float z = (float)iz + 0.5f;

    if (ix < imageW && iy < imageH && iz < imageD)
    {
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)
                for (float k = -NLM_WINDOW_RADIUS; k <= NLM_WINDOW_RADIUS; k++)
                {
                    //Find color distance from (x, y) to (x + j, y + i)
                    float weightIJK = 0;

                    for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                        for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++)
                            for (float l = -NLM_BLOCK_RADIUS; l <= NLM_BLOCK_RADIUS; l++){
                                float k1 = tex3D(nlmTex, x + j + m, y + i + n,z+l+k);
                                float k2 = tex3D(nlmTex, x + m, y + n,z+l);

                                //weightIJK += k1*logf(k1)+k2*logf(k2)-(k1+k2)*logf((k1+k2)*0.5f);
                                float test = k1*logf(k1)+k2*logf(k2)-(k1+k2)*logf((k1+k2)*0.5f);
                                //float test = (k1-k2)*(k1-k2)/(k1+k2);
                                if (!isnan(test)) weightIJK += test;
                            }




                    //Derive final weight from color and geometric distance
                    weightIJK     = expf(-(weightIJK * Noise ));
                    //weightIJK     = expf(-(weightIJK * Noise ));




                    //Accumulate (x + j, y + i) texel color with computed weight#
                    float IJK = tex3D(nlmTex, x + j, y + i,z+k);
                    accum += IJK * weightIJK;


                    //Sum of weights for color normalization to [0..1] range
                    sumWeights  += weightIJK;

                    //Update weight counter, if NLM weight for current window texel
                    //exceeds the weight threshold
                    //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
                }

        //Normalize result color by sum of weights

        accum /= sumWeights;

        //if (fCount > NLM_LERP_THRESHOLD)
        dst[imageW * iy + ix+imageH*imageW*iz] = accum ;
    }
}
__global__ static void NLM2D(
        float *dst,
        int imageW,
        int imageH,
        float Noise
)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;


    if (ix < imageW && iy < imageH)
    {
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)

            {
                //Find color distance from (x, y) to (x + j, y + i)
                float weightIJK = 0;

                for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                    for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++){

                        float diff =tex2D(nlmTex2D, x + j + m, y + i + n) -
                                    tex2D(nlmTex2D, x + m, y + n);
                        weightIJK += diff*diff;
                    }




                //Derive final weight from color and geometric distance
                weightIJK     = expf(-(weightIJK * Noise + (i * i + j * j) * INV_NLM_WINDOW_AREA2D));
                //weightIJK     = expf(-(weightIJK * Noise ));




                //Accumulate (x + j, y + i) texel color with computed weight#
                float IJK = tex2D(nlmTex2D, x + j, y + i);
                accum += IJK * weightIJK;


                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJK;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights

        accum /= sumWeights;

        //if (fCount > NLM_LERP_THRESHOLD)
        dst[imageW * iy + ix] = accum ;
    }
}
__global__ static void NLM2DPoisson(
        float *dst,
        int imageW,
        int imageH,
        float Noise
)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    //Add half of a texel to always address exact texel centers
    const float x = (float)ix + 0.5f;
    const float y = (float)iy + 0.5f;


    if (ix < imageW && iy < imageH)
    {
        //Normalized counter for the NLM weight threshold
        //float fCount = 0;
        //Total sum of pixel weights
        float sumWeights = 0;
        //Result accumulator
        float accum = 0;

        //Cycle through NLM window, surrounding (x, y) texel
        for (float i = -NLM_WINDOW_RADIUS; i <= NLM_WINDOW_RADIUS; i++)
            for (float j = -NLM_WINDOW_RADIUS; j <= NLM_WINDOW_RADIUS; j++)

            {
                //Find color distance from (x, y) to (x + j, y + i)
                float weightIJK = 0;

                for (float n = -NLM_BLOCK_RADIUS; n <= NLM_BLOCK_RADIUS; n++)
                    for (float m = -NLM_BLOCK_RADIUS; m <= NLM_BLOCK_RADIUS; m++){

                        float k1 = tex2D(nlmTex2D, x + j + m, y + i + n);
                        float k2 = tex2D(nlmTex2D, x + m, y + n);

                        //weightIJK += k1*logf(k1)+k2*logf(k2)-(k1+k2)*logf((k1+k2)*0.5f);
                        float test = k1*logf(k1)+k2*logf(k2)-(k1+k2)*logf((k1+k2)*0.5f);
                        //float test = (k1-k2)*(k1-k2)/(k1+k2);
                        if (!isnan(test)) weightIJK += test;

                    }




                //Derive final weight from color and geometric distance
                weightIJK     = expf(-(weightIJK * Noise) );
                //weightIJK     = expf(-(weightIJK * Noise ));




                //Accumulate (x + j, y + i) texel color with computed weight#
                float IJK = tex2D(nlmTex2D, x + j, y + i);
                accum += IJK * weightIJK;


                //Sum of weights for color normalization to [0..1] range
                sumWeights  += weightIJK;

                //Update weight counter, if NLM weight for current window texel
                //exceeds the weight threshold
                //fCount      += (weightIJ > NLM_WEIGHT_THRESHOLD) ? INV_NLM_WINDOW_AREA : 0;
            }

        //Normalize result color by sum of weights
        if (sumWeights > 0)
            accum /= sumWeights;

        //if (fCount > NLM_LERP_THRESHOLD)
        dst[imageW * iy + ix] = accum ;
    }
}


void Gadgetron::nonlocal_means_block(
        cuNDArray<float> *input, cuNDArray<float> *output ,
        float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    nlmTex.addressMode[0] = hipAddressModeClamp;
    nlmTex.addressMode[1] = hipAddressModeClamp;
    nlmTex.addressMode[2] = hipAddressModeClamp;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();
    hipExtent extent;
    extent.width = imageW;
    extent.height = imageH;
    extent.depth = imageD;

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.kind = hipMemcpyDeviceToDevice;
    cpy_params.extent = extent;

    hipArray *image_array;
    hipMalloc3DArray(&image_array, &channelDesc, extent);
    cpy_params.dstArray = image_array;
    cpy_params.srcPtr = make_hipPitchedPtr
            ((void *) input->get_data_ptr(), extent.width * sizeof(float), extent.width, extent.height);
    hipMemcpy3D(&cpy_params);

    hipBindTextureToArray(nlmTex, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y,BLOCKDIM_Z);

    dim3 grid((imageW+threads.x-1)/threads.x, (imageH+threads.y-1)/threads.y,(imageD+threads.z-1)/threads.z);

    for (int offset =0; offset < BLOCKDIM_X/2; offset++) {
        NLM3DBLOCK << < grid, threads >> >
                              (output->get_data_ptr(), imageW, imageH, imageD, offset,1.0 / (Noise * Noise));

        hipDeviceSynchronize();
    }

    hipFreeArray(image_array);
}


void Gadgetron::nonlocal_meansPoisson(
        cuNDArray<float> *input, cuNDArray<float> *output ,
        float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    nlmTex.addressMode[0] = hipAddressModeClamp;
    nlmTex.addressMode[1] = hipAddressModeClamp;
    nlmTex.addressMode[2] = hipAddressModeClamp;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();
    hipExtent extent;
    extent.width = imageW;
    extent.height = imageH;
    extent.depth = imageD;

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.kind = hipMemcpyDeviceToDevice;
    cpy_params.extent = extent;

    hipArray *image_array;
    hipMalloc3DArray(&image_array, &channelDesc, extent);
    cpy_params.dstArray = image_array;
    cpy_params.srcPtr = make_hipPitchedPtr
            ((void *) input->get_data_ptr(), extent.width * sizeof(float), extent.width, extent.height);
    hipMemcpy3D(&cpy_params);

    hipBindTextureToArray(nlmTex, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y,BLOCKDIM_Z);

    dim3 grid((imageW+threads.x-1)/threads.x, (imageH+threads.y-1)/threads.y,(imageD+threads.z-1)/threads.z);

    NLM3DPoisson<<<grid, threads>>>(output->get_data_ptr(), imageW, imageH,imageD, 1.0/(Noise*Noise));
    hipFreeArray(image_array);
}

void Gadgetron::nonlocal_means(
    cuNDArray<float> *input, cuNDArray<float> *output ,
    float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    nlmTex.addressMode[0] = hipAddressModeClamp;
    nlmTex.addressMode[1] = hipAddressModeClamp;
    nlmTex.addressMode[2] = hipAddressModeClamp;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();
    hipExtent extent;
    extent.width = imageW;
    extent.height = imageH;
    extent.depth = imageD;

    hipMemcpy3DParms cpy_params = {0};
    cpy_params.kind = hipMemcpyDeviceToDevice;
    cpy_params.extent = extent;

    hipArray *image_array;
    hipMalloc3DArray(&image_array, &channelDesc, extent);
    cpy_params.dstArray = image_array;
    cpy_params.srcPtr = make_hipPitchedPtr
            ((void *) input->get_data_ptr(), extent.width * sizeof(float), extent.width, extent.height);
    hipMemcpy3D(&cpy_params);

    hipBindTextureToArray(nlmTex, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y,BLOCKDIM_Z);

    dim3 grid((imageW+threads.x-1)/threads.x, (imageH+threads.y-1)/threads.y,(imageD+threads.z-1)/threads.z);

    NLM3D<<<grid, threads>>>(output->get_data_ptr(), imageW, imageH,imageD, 1.0/(Noise*Noise));
    hipFreeArray(image_array);
}

void Gadgetron::nonlocal_means2D(
        cuNDArray<float> *input, cuNDArray<float> *output ,
        float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    nlmTex2D.addressMode[0] = hipAddressModeClamp;
    nlmTex2D.addressMode[1] = hipAddressModeClamp;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();

    hipArray *image_array;
    hipMallocArray(&image_array, &channelDesc, imageW, imageH);

    float* input_ptr = input->get_data_ptr();
    float* output_ptr = output->get_data_ptr();

    std::vector<size_t> dims2D = {imageW,imageH};
    for (int i = 0; i < imageD; i++) {


        cuNDArray<float> input_view(dims2D,input_ptr);

        hipMemcpyToArray(image_array, 0, 0, input_view.get_data_ptr(), input_view.get_number_of_bytes(),
                          hipMemcpyDeviceToDevice);

        hipBindTextureToArray(nlmTex2D, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


        dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);

        dim3 grid((imageW + threads.x - 1) / threads.x, (imageH + threads.y - 1) / threads.y);

        NLM2D << < grid, threads >> > (output_ptr, imageW, imageH, 1.0 / (Noise * Noise));

        output_ptr += input_view.get_number_of_elements();
        input_ptr += input_view.get_number_of_elements();


    }
    hipFreeArray(image_array);
}



void Gadgetron::nonlocal_means2DPoisson(
        cuNDArray<float> *input, cuNDArray<float> *output ,
        float Noise
)
{
    if (!input->dimensions_equal(output))
        throw std::runtime_error("Input and output dimensions must agree");

    int imageW = input->get_size(0);
    int imageH = input->get_size(1);
    int imageD = input->get_size(2);


    nlmTex2D.addressMode[0] = hipAddressModeClamp;
    nlmTex2D.addressMode[1] = hipAddressModeClamp;

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < float > ();

    hipArray *image_array;
    hipMallocArray(&image_array, &channelDesc, imageW, imageH);

    float* input_ptr = input->get_data_ptr();
    float* output_ptr = output->get_data_ptr();

    std::vector<size_t> dims2D = {imageW,imageH};
    for (int i = 0; i < imageD; i++) {


        cuNDArray<float> input_view(dims2D,input_ptr);

        hipMemcpyToArray(image_array, 0, 0, input_view.get_data_ptr(), input_view.get_number_of_bytes(),
                          hipMemcpyDeviceToDevice);

        hipBindTextureToArray(nlmTex2D, image_array, channelDesc);CHECK_FOR_CUDA_ERROR();


        dim3 threads(BLOCKDIM_X, BLOCKDIM_Y, BLOCKDIM_Z);

        dim3 grid((imageW + threads.x - 1) / threads.x, (imageH + threads.y - 1) / threads.y);

        NLM2DPoisson << < grid, threads >> > (output_ptr, imageW, imageH, 1.0 / (Noise * Noise));

        output_ptr += input_view.get_number_of_elements();
        input_ptr += input_view.get_number_of_elements();


    }
    hipFreeArray(image_array);
}
